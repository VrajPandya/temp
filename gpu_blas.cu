#include <hipblas.h>
#include <hiprand.h>
#include <hip/hip_runtime.h>
#include "gpu_blas.h"

void gpu_sgemm(int m, int n, int k, float alpha, float* h_A, int lda,
		float* h_B, int ldb, float beta, float* h_C, int ldc) {
	// Create a handle for CUBLAS
	hipblasHandle_t handle;
	hipblasCreate(&handle);

	// Fill the arrays A and B on GPU with random numbers
	float *d_A, *d_B, *d_C;
	hipMalloc(&d_A, m * n * sizeof(float));
	hipMalloc(&d_B, n * k * sizeof(float));
	hipMalloc(&d_C, m * k * sizeof(float));

	// If you already have useful values in A and B you can copy them in GPU:
	hipMemcpy(d_A, h_A, m * n * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, n * m * sizeof(float), hipMemcpyHostToDevice);

	// Do the actual multiplication
	hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, d_A, lda,
			d_B, ldb, &beta, d_C, ldc);

	// Copy (and print) the result on host memory
	hipMemcpy(h_C, d_C, m * k * sizeof(float), hipMemcpyDeviceToHost);

	// Destroy the handle
	hipblasDestroy(handle);
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
}
