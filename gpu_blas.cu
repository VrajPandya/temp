#include <hipblas.h>
#include <hiprand.h>
#include <hip/hip_runtime.h>
#include "gpu_blas.h"

void gpu_sgemm(int m, int n, int k, float alpha, float* h_A, int lda,
		float* h_B, int ldb, float beta, float* h_C, int ldc) {
	// Create a handle for CUBLAS
	hipblasHandle_t handle;
	hipblasCreate(&handle);

	// Fill the arrays A and B on GPU with random numbers
	float *d_A, *d_B, *d_C;
	hipMalloc(&d_A, m * n * sizeof(float));
	hipMalloc(&d_B, n * k * sizeof(float));
	hipMalloc(&d_C, m * k * sizeof(float));

	// If you already have useful values in A and B you can copy them in GPU:
	hipMemcpy(d_A, h_A, m * n * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, n * m * sizeof(float), hipMemcpyHostToDevice);

	// Do the actual multiplication
	hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, d_A, lda,
			d_B, ldb, &beta, d_C, ldc);

	// Copy (and print) the result on host memory
	hipMemcpy(h_C, d_C, m * k * sizeof(float), hipMemcpyDeviceToHost);

	// Destroy the handle
	hipblasDestroy(handle);
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
}

void gpu_dgemm(int m, int n, int k, double alpha, double* h_A, int lda,
		double* h_B, int ldb, double beta, double* h_C, int ldc) {
	// Create a handle for CUBLAS
	hipblasHandle_t handle;
	hipblasCreate(&handle);


	double *d_A, *d_B, *d_C;
	hipMalloc(&d_A, m * n * sizeof(double));
	hipMalloc(&d_B, n * k * sizeof(double));
	hipMalloc(&d_C, m * k * sizeof(double));


	hipMemcpy(d_A, h_A, m * n * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, n * m * sizeof(double), hipMemcpyHostToDevice);

	// Do the actual multiplication
	hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, d_A, lda,
			d_B, ldb, &beta, d_C, ldc);


	hipMemcpy(h_C, d_C, m * k * sizeof(double), hipMemcpyDeviceToHost);

	// Destroy the handle
	hipblasDestroy(handle);
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
}
